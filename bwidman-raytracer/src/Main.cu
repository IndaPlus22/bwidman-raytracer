#include "hip/hip_runtime.h"
﻿// Simple GPU accelerated ray tracer
// Author: Benjamin Widman (benjaneb)
#define __HIPCC__
#include <GLFW/glfw3.h>

#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"

#include <iostream>

#include "Math.cuh"
#include "WorldTypes.cuh"
#include "Controls.cuh"

#define WIDTH 1280
#define HEIGHT 720

unsigned int screenTexture;
hipGraphicsResource_t cudaImage; // Must be global

scene allocateScene() {
    camera hCamera = { ZERO_VEC, { { 0, 0, 1 }, { 1, 0, 0 } }, 0, 0, PI / 2 };
    sphere hSpheres[] = {
        // Position, radius, color
        { { 2, 0, 8 }, 2, { 200, 50, 0 } },
        { { -2, 0, 6 }, 1, { 50, 0, 200 } },
    };
    int hSphereCount = sizeof(hSpheres) / sizeof(sphere);

    // Allocate spheres on GPU
    sphere* dSpheres;
    hipMalloc(&dSpheres, sizeof(hSpheres));
    hipMemcpy(dSpheres, &hSpheres, sizeof(hSpheres), hipMemcpyHostToDevice);

    return { hCamera, dSpheres, hSphereCount };
}

// Check if camera ray intersects with sphere
__device__ bool sphereIntersect(ray cameraRay, sphere sphere, color* pixel, vec3d* intersection, float* closestHit) {
    // If you don't have the tex comments extension, good luck reading this
    //tex:
    // Sphere equation:
    // $$(x-p_1)^2 + (y-p_2)^2 + (z-p_3)^2 = r^2$$
    // Ray equation: $$\vec{r} = \vec{x} + t\vec{v}$$
    // Input ray into sphere equation:
    // $$(x_1+tv_1-p_1)^2 + (x_2+tv_2-p_2)^2 + (x_3+tv_3-p_3)^2 = r^2$$
    // $$(x_1^2 + x_1v_1t - x_1p_1 + x_1v_1t + v_1^2t^2 - v_1p_1t - p_1x_1 - p_1v_1t + p_1^2) + ... - r^2 = 0$$
    // $$(v_1^2 + v_2^2 + v_3^2)t^2 + (2x_1v_1 - 2p_1v_1 + 2x_2v_2 - 2p_2v_2 + 2x_3v_3 - 2p_3v_3)t + (x_1^2 - 2p_1x_1 + p_1^2 + x_2^2 - 2p_2x_2 + p_2^2 + x_3^2 - 2p_3x_3 + p_3^2 - r^2) = 0$$
    // $$(v_1^2 + v_2^2 + v_3^2)t^2 + 2(x_1v_1 - p_1v_1 + x_2v_2 - p_2v_2 + x_3v_3 - p_3v_3)t + ((x_1 - p_1)^2 + (x_2 - p_2)^2 + (x_3 - p_3)^2 - r^2) = 0$$
    // $$(\vec{v} \cdot \vec{v})t^2 + 2((\vec{x} - \vec{p}) \cdot \vec{v})t + ((\vec{x} - \vec{p}) \cdot (\vec{x} - \vec{p}) - r^2) = 0$$
    // Solve for t with quadratic formula:
    // $$t = \frac{-b\pm\sqrt{b^2 - 4ac}}{2a}$$
    // Where:
    // $$a = \vec{v} \cdot \vec{v}$$
    // $$b = 2((\vec{x} - \vec{p}) \cdot \vec{v})$$
    // $$c = (\vec{x} - \vec{p}) \cdot (\vec{x} - \vec{p}) - r^2$$
    vec3d p = sphere.position;
    vec3d x = cameraRay.origin;
    vec3d v = cameraRay.direction;

    float a = dotProduct(v, v);
    float b = 2 * dotProduct(x - p, v);
    float c = dotProduct(x - p, x - p) - sphere.radius * sphere.radius;

    float discriminant = b * b - 4 * a * c; // Discriminator of all

    // Negative root => no solutions => no intersection
    if (discriminant < 0) {
        return false;
    }

    // Only interested in negative solution to the root as it gives the
    // smallest value of t and is therefore the closest to the camera
    float t = (-b - sqrtf(discriminant)) / (2 * a);

    // Behind camera or further away than the so far closest hit
    if (t < 0 || t > *closestHit) {
        return false;
    }

    *closestHit = t;
    *pixel = sphere.color;
    *intersection = cameraRay.origin + t * cameraRay.direction;
    return true;
}

__device__ color raytrace(ray cameraRay, sphere spheres[], int sphereCount) {
    color pixel = ZERO_VEC;
    float closestHit = INFINITY; // Gets updated for every new closest hit
    
    // Check intersection with all spheres
    for (int i = 0; i < sphereCount; i++) {
        vec3d intersection;
        bool intersected = sphereIntersect(cameraRay, spheres[i], &pixel, &intersection, &closestHit);
    }

    return pixel;
}

__global__ void launch_raytracer(hipSurfaceObject_t screenSurfaceObj, dim3 cell, scene scene, float screenZ, matrix3d rotLeft, matrix3d rotUp) {
    int pixelStartX = (blockIdx.x * blockDim.x + threadIdx.x) * cell.x;
    int pixelStartY = (blockIdx.y * blockDim.y + threadIdx.y) * cell.y;

    // Loop through pixels in designated screen cell
    for (int y = 0; y < cell.y; y++) {
        for (int x = 0; x < cell.x; x++) {
            float screenX = pixelStartX + x;
            float screenY = pixelStartY + y;

            vec3d pixelPosition = { screenX - WIDTH / 2, screenY - HEIGHT / 2, screenZ };
            pixelPosition = rotLeft * rotUp * pixelPosition; // Rotate to camera's facing direction

            ray cameraRay = { scene.camera.position, normalize(pixelPosition) };

            color pixel = raytrace(cameraRay, scene.spheres, scene.sphereCount);

            surf2Dwrite(make_uchar4(pixel.r, pixel.g, pixel.b, 255), screenSurfaceObj, screenX * sizeof(uchar4), screenY);
        }
    }
}

void render(scene scene) {
    hipError_t error;
    error = hipGraphicsMapResources(1, &cudaImage);

    // Map texture array to cuda
    hipArray_t screenCudaArray;
    error = hipGraphicsSubResourceGetMappedArray(&screenCudaArray, cudaImage, 0, 0);

    // Data describing array
    hipResourceDesc screenArrayDesc;
    screenArrayDesc.resType = hipResourceTypeArray;
    screenArrayDesc.res.array.array = screenCudaArray;

    // Create read-/writeable object for screen array
    hipSurfaceObject_t screenSurfaceObj;
    error = hipCreateSurfaceObject(&screenSurfaceObj, &screenArrayDesc);
    if (error != hipSuccess)
        std::cout << "Failed to map screen array to cuda" << std::endl;

    // Screen coordinate calculations
    float screenZ = (WIDTH / 2) / tanf(scene.camera.FOV / 2);
    matrix3d rotLeft = rotationMatrix3DY(scene.camera.angle[0]);
    matrix3d rotUp = rotationMatrix3DX(scene.camera.angle[1]);

    // Calculate number of threads etc.
    //tex:Number of blocks (number of pixels / (threads/block * pixels/thread)):
    //$$\frac{1280*720}{256*9} = 400$$
    //$$400 = 20*20$$
    //(both 1280 and 720 are divisible by 20)
    
    // Hence, we have a 2D grid of 20 * 20 blocks consisting of 256 threads each (good number)
    // Each block handles a 64 * 36 pixel area
    // Each thread then handles a 1 * 9 pixel area
    dim3 grid(20, 20); // 20 * 20 blocks
    dim3 block(64, 4); // 64 * 4 threads
    dim3 cell(1, 9); // Dimensions of pixel cell handled by each thread

    launch_raytracer<<<grid, block>>>(screenSurfaceObj, cell, scene, screenZ, rotLeft, rotUp);

    // Clean up cuda objects
    error = hipDestroySurfaceObject(screenSurfaceObj);
    error = hipGraphicsUnmapResources(1, &cudaImage);

    error = hipStreamSynchronize(0); // Synchronize cuda stream 0 (the only one in use)
    if (error != hipSuccess)
        std::cout << "Failed to clean up cuda objects and/or synchronize" << std::endl;

    // Draw screen texture
    glBindTexture(GL_TEXTURE_2D, screenTexture);
    glBegin(GL_QUADS);

    // Texture coordinates    Vertex coordinates on screen
    glTexCoord2f(0.0f, 0.0f); glVertex2f(-1.0f, -1.0f);
    glTexCoord2f(1.0f, 0.0f); glVertex2f(1.0f, -1.0f);
    glTexCoord2f(1.0f, 1.0f); glVertex2f(1.0f, 1.0f);
    glTexCoord2f(0.0f, 1.0f); glVertex2f(-1.0f, 1.0f);

    glEnd();
    glBindTexture(GL_TEXTURE_2D, 0); // Unbind texture
    glFinish();
}

int main() {
    GLFWwindow* window;

    // Initialize GLFW
    if (!glfwInit()) {
        std::cout << "Failed to initialize GLFW!" << std::endl;
        std::cin.get();
        return -1;
    }

    // Create a windowed mode window and its OpenGL context
    window = glfwCreateWindow(WIDTH, HEIGHT, "bwidman-raytracer", NULL, NULL);
    if (!window) {
        glfwTerminate();
        return -1;
    }

    // Make the window's context current
    glfwMakeContextCurrent(window);

    // Set up OpenGL screen texture

    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &screenTexture);

    // Texture settings (rendering canvas)
    glBindTexture(GL_TEXTURE_2D, screenTexture);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, WIDTH, HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);

    glBindTexture(GL_TEXTURE_2D, 0);

    // Register texture and pixel buffer to cuda
    hipError_t error = hipGraphicsGLRegisterImage(&cudaImage, screenTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    if (error != hipSuccess) {
        std::cout << "Failed to register screen texture to cuda" << std::endl;
    }

    // Allocate the scene on the GPU
    scene scene = allocateScene();

    double deltaTime = 0;
    int frameCount = 0;
    // Loop until the user closes the window
    while (!glfwWindowShouldClose(window)) {
        // Start timer
        double startTime = glfwGetTime();

        // Render here
        glClear(GL_COLOR_BUFFER_BIT);

        render(scene);

        // Swap front and back buffers
        glfwSwapBuffers(window);

        controls(window, scene.camera);

        // Poll for and process events
        glfwPollEvents();

        // Stop timer and print FPS if over a second has elapsed since last print
        deltaTime += glfwGetTime() - startTime;
        frameCount++;
        if (deltaTime > 1.0) {
            std::cout << "FPS: " << frameCount / deltaTime << std::endl;
            deltaTime = 0; frameCount = 0;
        }
    }

    glfwTerminate();

    // Clean up scene
    hipFree(scene.spheres);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}